#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/scan.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdio.h>

namespace cg = cooperative_groups;


namespace
{
    constexpr unsigned int WarpSize = 4;


    template<typename T, typename F>
    __device__ void ExclusiveScan_block_kernel(const cg::thread_block& block, T* const pData, F&& op, T* const pBlockSum)
    {
        // 1. Allocate shared memory to hold intermediate results obtained from each warp in the thread block.
        // 2. The allocation size is conservative - the exact calculation is block.num_threads() / WarpSize.
        __shared__ T shared[WarpSize];

        const cg::thread_block_tile<WarpSize, cg::thread_block> warp = cg::tiled_partition<WarpSize, cg::thread_block>(block);

        // This ensures we can scan the intermediate results with only 1 warp.
        assert(warp.meta_group_rank() < WarpSize);

        const T originalVal = pData[block.thread_rank()];
        const T scannedVal  = cg::exclusive_scan(warp, originalVal, op);

        printf("Scanned val: %u\n", scannedVal);

        if (warp.thread_rank() + 1 == warp.num_threads())
        {
            shared[warp.meta_group_rank()] = scannedVal + originalVal;
        }

        block.sync();

        if ((block.thread_rank() + 1 == block.num_threads()) && pBlockSum)
        {
            *pBlockSum = scannedVal + originalVal;
        }

        if (warp.meta_group_rank() == 0)
        {
            shared[warp.thread_rank()] = cg::exclusive_scan(warp, shared[warp.thread_rank()], op);
        }

        block.sync();

        pData[block.thread_rank()] = shared[warp.meta_group_rank()] + scannedVal;
    }


    template<typename T>
    __global__ void ExclusiveScan_Add_grid_kernel(T* const pData, T* const pBlockSums)
    {
        const cg::grid_group   grid  = cg::this_grid();
        const cg::thread_block block = cg::this_thread_block();

        ExclusiveScan_block_kernel(block,
                                   &pData[grid.block_rank() * block.num_threads()],
                                   cg::plus<T>(),
                                   pBlockSums ? &pBlockSums[grid.block_rank()] : pBlockSums);
    }
}


int main()
{
    unsigned int data[12] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12};
    unsigned int blockSums[3];

    unsigned int* pDataDevice = nullptr;
    unsigned int* pBlockSums  = nullptr;
    hipMalloc(&pDataDevice, sizeof(data));
    hipMalloc(&pBlockSums, sizeof(blockSums));

    hipMemcpy(pDataDevice, data, sizeof(data), hipMemcpyHostToDevice);

    ExclusiveScan_Add_grid_kernel<<<3, 4>>>(pDataDevice, pBlockSums);
    ExclusiveScan_Add_grid_kernel<<<1, 4>>>(pBlockSums, (unsigned int*)0);

    hipMemcpy(data, pDataDevice, sizeof(data), hipMemcpyDeviceToHost);
    hipMemcpy(blockSums, pBlockSums, sizeof(blockSums), hipMemcpyDeviceToHost);

    for (const auto i : data)
    {
        std::cout << i << " ";
    }
    std::cout << std::endl;

    for (const auto i : blockSums)
    {
        std::cout << i << " ";
    }
    std::cout << std::endl;
}
