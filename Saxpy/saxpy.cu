#include "hip/hip_runtime.h"
#include "debug.h"
#include "saxpy.h"

#include ""

#include <algorithm>


__global__ void Saxpy(const float                     a,
                      const float* const __restrict__ pXDevice,
                      const float* const __restrict__ pYDevice,
                            float* const __restrict__ pZDevice,
                      const size_t                    len)
{
    unsigned int gridThreadIdx = (blockDim.x * blockIdx.x) + threadIdx.x;
    unsigned int gridSize      = gridDim.x * blockDim.x;

    for (size_t idx = gridThreadIdx; idx < len; idx += gridSize)
    {
        pZDevice[idx] = (a * pXDevice[idx]) + pYDevice[idx];
    }
}


void saxpy::DeviceExecute(const float        a,
                          const float* const pXDevice,
                          const float* const pYDevice,
                                float* const pZDevice,
                          const size_t       len,
                          const hipStream_t stream)
{
    if (len > 0)
    {
        // NOTE: these values require hw-specific tuning.
        const unsigned int tpb           = 128;
        const size_t       maxAllowedBpg = 256;
        const size_t       maxNeededBpg  = (len + tpb - 1) / tpb;
        const unsigned int bpg           = static_cast<unsigned int>(std::min(maxAllowedBpg, maxNeededBpg));

        DBG_MSG_STD_OUT("Saxpy launch parameters:\n\tLen: ", len, "\n\tTPB: ", tpb, "\n\tBPG: ", bpg);

        Saxpy<<<bpg, tpb, 0, stream>>>(a, pXDevice, pYDevice, pZDevice, len);
    }
}


void saxpy::HostExecute(const float        a,
                        const float* const pXHost,
                        const float* const pYHost,
                              float* const pZHost,
                        const size_t len)
{
    std::transform(pXHost, pXHost + len,
                   pYHost,
                   pZHost,
                   [=](float x, float y) { return (a * x) + y; });
}